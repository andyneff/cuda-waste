#include "hip/hip_runtime.h"
#include <iostream>

__global__ void fun(float * d, int size)
{
	extern __shared__ int buf[];
    int idx = threadIdx.x +
            blockDim.x * blockIdx.x
            + blockDim.x * gridDim.x * blockDim.y * blockIdx.y
            + blockDim.x * gridDim.x * threadIdx.y;
    if (idx < 0)
        return;
    if (idx >= size)
        return;
	buf[idx] = idx;
	__syncthreads();
    d[idx] = buf[(idx+50) % size] * 10.0 / 0.1;
}

int main()
{
    float * h;
    float * d;
    int size = 100;
	int shared = 400;
    h = (float*)malloc(size*sizeof(float));
    hipMalloc(&d, size*sizeof(float));
    hipMemcpy(d, h, size*sizeof(float), hipMemcpyHostToDevice);
    fun<<<1, size, shared>>>(d, size);
    hipDeviceSynchronize();
    int rv = hipGetLastError();
    hipMemcpy(h, d, size*sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < size; ++i)
        std::cout << "Result = " << h[i] << "\n";
    return 0;
}
