#include "hip/hip_runtime.h"
#include <iostream>

__global__ void fun(int * d, int * e)
{
	*d = 1;
	*e = 2;
}

int main()
{
	int h = 0;
	int * d;
	int * e;
	hipMalloc(&d, sizeof(int));
	hipMalloc(&e, sizeof(int));
	hipMemcpy(d, &h, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(e, &h, sizeof(int), hipMemcpyHostToDevice);
	fun<<<1,1>>>(d, e);
	hipDeviceSynchronize();
	int rv = hipGetLastError();
	hipMemcpy(&h, d, sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "Result = " << h << "\n";
	hipMemcpy(&h, e, sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "Result = " << h << "\n";
	return 0;
}
