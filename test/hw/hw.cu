#include "hip/hip_runtime.h"
#include <iostream>

__global__ void fun(int * mem)
{
	*mem = 1;
}

int main()
{
	int h = 0;
	int * d;
	hipMalloc(&d, sizeof(int));
	hipMemcpy(d, &h, sizeof(int), hipMemcpyHostToDevice);
	fun<<<1,1>>>(d);
	hipDeviceSynchronize();
	int rv = hipGetLastError();
	hipMemcpy(&h, d, sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "Result = " << h << "\n";
	return 0;
}
