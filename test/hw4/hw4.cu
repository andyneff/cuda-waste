#include "hip/hip_runtime.h"
#include <iostream>

__global__ void fun(float * d, int size)
{
    int idx = threadIdx.x +
            blockDim.x * blockIdx.x
            + blockDim.x * gridDim.x * blockDim.y * blockIdx.y
            + blockDim.x * gridDim.x * threadIdx.y;
    if (idx < 0)
        return;
    if (idx >= size)
        return;
    d[idx] = idx * 10.0 / 0.1;
}

int main()
{
    float * h;
    float * d;
    int size = 100;
    h = (float*)malloc(size*sizeof(float));
    hipMalloc(&d, size*sizeof(float));
    hipMemcpy(d, h, size*sizeof(float), hipMemcpyHostToDevice);
    fun<<<1,size>>>(d, size);
    hipDeviceSynchronize();
    int rv = hipGetLastError();
    hipMemcpy(h, d, size*sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < size; ++i)
        std::cout << "Result = " << h[i] << "\n";
    return 0;
}
