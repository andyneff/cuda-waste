#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

void CUDA_CALL(hipError_t result)
{
	if (result != hipSuccess)
	{
		std::cout << "Error " << result << "\n";
		exit(1);
	}
}

texture<float, 1, hipReadModeElementType> texRef1;


// Simple transformation kernel
__global__ void transformKernel(float* output_x, float * output_y, int width, int height, int output_width, int output_height, float theta)
{
	// Calculate normalized texture coordinates
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
     
	float u = x / (float)width;
	float v = y / (float)height;

	float tu = ((float)width * x) / output_width;
	float tv = ((float)height * y) / output_height;

	output_x[y * output_width + x] = tu;
	output_y[y * output_width + x] = tex1D(texRef1, tu);
}

// Host code
int main()
{
	int width = 10;
	int height = 1;
	int size = width * height;

	float * h_data = (float*) malloc(width * height * sizeof(float));
	for (int i = 0 ; i < height; i++)
		for (int j = 0; j < width; j++)
				h_data[i * width + j] = i * width + j + 1;

	printf ("\n Original array \n");
	for (int i = 0; i < height; i++)
	{
		for (int j = 0 ; j < width; j++)
			printf ("%f ", h_data[i*width + j]);
		printf ("\n");
	}
          
	// create a CUDA array on the device
	hipArray* cuArray;
	CUDA_CALL( hipMallocArray(&cuArray, &texRef1.channelDesc, size, height) );
	CUDA_CALL( hipMemcpyToArray(cuArray, 0, 0, h_data, sizeof(float) * size, hipMemcpyHostToDevice) );
   
	// bind a texture to the CUDA array
	hipBindTextureToArray (texRef1, cuArray);
 
	// Set texture access
	texRef1.normalized = false;
	texRef1.filterMode = hipFilterModeLinear;
        
	int output_width = 100;
	int output_height = 1;

	// Set up block dims.
	dim3 dimBlock(10, 1);
	dim3 dimGrid((output_width - 1)/dimBlock.x + 1, (output_height - 1)/dimBlock.y + 1);
     
	printf("blockDim = %d, %d\n", dimBlock.x, dimBlock.y);
	printf("gridDim = %d, %d\n", dimGrid.x, dimGrid.y);

	// Allocate result of transformation in device memory
	int output_size =  dimGrid.x * dimBlock.x * dimGrid.y * dimBlock.y;
	printf("output_size %d\n", output_size);
	float* d_output_x;
	float* d_output_y;
	CUDA_CALL( hipMalloc(&d_output_x, output_size * sizeof(float)) );
	CUDA_CALL( hipMalloc(&d_output_y, output_size * sizeof(float)) );
	float * h_output_x = (float*)malloc(output_size * sizeof(float));
	float * h_output_y = (float*)malloc(output_size * sizeof(float));

	transformKernel<<<dimGrid, dimBlock>>>(d_output_x, d_output_y, width, height, output_width, output_height, 90);
    hipDeviceSynchronize();

	CUDA_CALL( hipMemcpy(h_output_x, d_output_x, output_size * sizeof(float), hipMemcpyDeviceToHost) );
	CUDA_CALL( hipMemcpy(h_output_y, d_output_y, output_size * sizeof(float), hipMemcpyDeviceToHost) );

	printf ("\nAfter operation\n");
	for (int i = 0; i < output_height; i++)
	{
		for (int j = 0; j < output_width; j++)
			printf ("%f %f\n", h_output_x[i*output_width + j], h_output_y[i*output_width + j]);
		printf ("\n");
	}

	system ("pause");
	// Free device memory
	hipFree(d_output_x);
	hipFree(d_output_y);
	return 0;
}

