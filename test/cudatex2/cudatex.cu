#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

void CUDA_CALL(hipError_t result)
{
    if (result != hipSuccess)
    {
        std::cout << "Error " << result << "\n";
		std::cout << hipGetErrorString(result) << "\n";
        exit(1);
    }
}

texture<float, hipTextureType1D, hipReadModeElementType> texRef;
texture<float, 2> texRef2;


// Simple transformation kernel
__global__ void transformKernel(float* output, int width, int height, int output_width, int output_height, float theta)
{
    // Calculate normalized texture coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    float u = x / (float)width;
    float v = y / (float)height;

    float tu = ((float)width * x) / output_width;
    float tv = ((float)height * y) / output_height;
    output[y * output_width + x] = tex2D(texRef2, tu, tv);
}

// Host code
int main()
{
    int width = 32;
    int height = 16;
    int size = width * height;

    float * h_data = (float*) malloc(width * height * sizeof(float));
    for (int i = 0 ; i < height; i++)
        for (int j = 0; j < width; j++)
                h_data[i * width + j] = i * width + j + 1;

    printf ("\n Original array \n");
    for (int i = 0; i < height; i++)
    {
        for (int j = 0 ; j < width; j++)
            printf ("%f ", h_data[i*width + j]);
        printf ("\n");
    }

          
    float * d_data;
    CUDA_CALL( hipMalloc(&d_data, size * sizeof(float)) );
    // Copy to device memory some data located at address h_data
    // in host memory
    CUDA_CALL( hipMemcpy(d_data, h_data, size * sizeof(float), hipMemcpyHostToDevice) );
    
    // Set texture parameters
    //texRef.addressMode[0] = hipAddressModeWrap;
    //texRef.addressMode[1] = hipAddressModeWrap;
    //texRef.filterMode = hipFilterModeLinear;
    //texRef.normalized = true;
     
    // Bind the array to the texture reference
	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

    CUDA_CALL( hipBindTexture2D(0, texRef2, d_data, desc, width, height, sizeof(float) * width) );
    hipDeviceSynchronize();
    
    int output_width = 32;
    int output_height = 16;

    // Set up block dims.
    dim3 dimBlock(4, 4);
    dim3 dimGrid((output_width - 1)/dimBlock.x + 1, (output_height - 1)/dimBlock.y + 1);
     
    printf("blockDim = %d, %d\n", dimBlock.x, dimBlock.y);
    printf("gridDim = %d, %d\n", dimGrid.x, dimGrid.y);

    // Allocate result of transformation in device memory
    int output_size =  dimGrid.x * dimBlock.x * dimGrid.y * dimBlock.y;
    printf("output_size %d\n", output_size);
    float* output;
    CUDA_CALL( hipMalloc(&output, output_size * sizeof(float)) );
    float * h_output = (float*)malloc(output_size * sizeof(float));
     
    transformKernel<<<dimGrid, dimBlock>>>(output, width, height, output_width, output_height, 90);
    hipDeviceSynchronize();
    CUDA_CALL( hipMemcpy(h_output, output, output_size * sizeof(float), hipMemcpyDeviceToHost) );

    printf ("\nAfter operation\n");
    for (int i = 0; i < output_height; i++)
    {
        for (int j = 0; j < output_width; j++)
            printf ("%d %f\n", i*output_width + j, h_output[i*output_width + j]);
        printf ("\n");
    }

    system ("pause");
    // Free device memory
    hipFree(d_data);
    hipFree(output);
    return 0;
}
